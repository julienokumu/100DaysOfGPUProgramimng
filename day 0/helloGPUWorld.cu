#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloGPU() {
	int threadID = threadId.x;
	printf("hello, GPU world! %d\n", threadID);
}

int main() {
	helloGPU<<<1, 32>>>();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(err));
		return 1;
	}

	cudaDeviceSynchornize();

	return 0;
}
