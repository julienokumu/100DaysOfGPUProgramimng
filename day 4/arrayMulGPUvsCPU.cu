#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void arrayMul(const float *A, const float *B, float *C, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	for (; idx < size; idx += blockDim.x * gridDim.x) {
		C[idx] = A[idx] * B[idx];
	}
}

void arrayMulCPU(const float *A, const float *B, float *C, int size) {
	for (int i = 0; i < size; i++) {
		C[i] = A[i] * B[i];
	}
}

int main() {
	const int N = 10000;
	size_t size = N * sizeof(float);
	const int blockSizes[] = {64, 128, 256};
	const int numTests = 3;

	float *h_A = (float *)malloc(size);
	float *h_B = (float *)malloc(size);
	float *h_C = (float *)malloc(size);
	float *h_C_cpu = (float *)malloc(size);

	for (int i = 0; i < N; i++) {
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	for (int t = 0; t < numTests; t++) {
		int threadsPerBlock = blockSizes[t];
		int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
		
		printf("\nTesting block size: %d (blocks: %d)\n", threadsPerBlock, blocks);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);

		arrayMul<<<blocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			printf("CUDA error: %s\n", hipGetErrorString(err));

			return 1;
		}

		hipEventRecord(stop);
		hipDeviceSynchronize(stop);

		float gpu_time = 0;
		hipEventElapsedTime(&gpu_time, start, stop);
		printf("GPU execution time: %.3f ms\n", gpu_time);

		hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

		auto cpu_start = std::chrono::high_resolution_clock::now();
		arrayMulCPU(h_A, h_B, h_C_cpu, N);
		auto cpu_end = std::chrono::high_resolution_clock::now();

		float cpu_time = std::chrono::duration<float, std::milli>(cpu_end - cpu_start).count();
		printf("CPU execution time : %.3f ms\n", cpu_time);

		bool correct = true;
		for (int i = 0; i < N; i++) {
			if (fabs(h_C[i] - h_C_cpu[i]) > 1e-5) {
				printf("verification failed at index %d: GPU = %.2f, CPU = %.2f\n", i, h_C[i], h_C_cpu[i]);

				correct = false;
				break;
			}
		}

		if (correct) {
			printf("multiplication completed successfully\n");
			printf("speedup: %.2fX\n", cpu_time / gpu_time);
		}

		hipEventDestroy(start);
		hipEventDestroy(stop);

	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_cpu);

	return 0;
}
