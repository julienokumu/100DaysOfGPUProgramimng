#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void arrayMul(const float *A, const float *B, float *C, int size) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	for (; idx < size; idx += blockDim.x * gridDim.x) {
		C[idx] = A[idx] * B[idx];
	}
}

int main () {
	const int N = 10000;
	size_t size = N * sizeof(float);

	const int blockSizes[] = {64, 128, 256};
	const int numTests = 3;

	float *h_A = (float *)malloc(size);
	float *h_B = (float *)malloc(size);
	float *h_C = (float *)malloc(size);

	for (int i = 0; i < N; i++) {
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	for (int t = 0; t < numTests; t++) {
		int threadsPerBlock = blockSizes[t];
		int blocks = (N - threadsPerBlock - 1) / threadsPerBlock;

		printf("\nTesting block size: %d (blocks: %d)\n", threadsPerBlock, blocks);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);

		arrayMul<<<threadsPerBlock, blocks>>>(d_A, d_B, d_C, N);

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			printf("CUDA error: %s\n", hipGetErrorString(err));

			return 1;
		}

		hipEventRecord(stop);
		hipEventSynchronize(stop);

		float gpu_time = 0;
		hipEventElapsedTime(&gpu_time, start, stop);
		printf("GPU execution time: %.3f ms\n", gpu_time);

		hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

		printf("First 5 results:\n");
		for (int i = 0; i < 5; i++) {
			printf("c[%d] = %.2f (a[%d] = %.2f * b[%d] = %.2f)\n", i, h_C[i], i, h_A[i], i, h_B[i]);
		}

		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
