#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dotProduct(const float *A, const float *B, float *C, int N) {
	__shared__ float sharedMem[256];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	float temp = 0.0f;
	if (idx < N) {
		temp = A[idx] * B[idx];
	}

	sharedMem[threadIdx.x] = temp;
	__syncthreads();

	for (int stride = blockDim.x / 2; stride > 0; stride >>=1) {
		if (threadIdx.x < stride) {
			sharedMem[threadIdx.x] += sharedMem[threadIdx.x + stride];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		atomicAdd(C, sharedMem[0]);
	}
}

void solve(const float *A, const float *B, float *C, int N) {
	const int threadsPerBlock = 256;
	int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

	hipMemset(C, 0, sizeof(float));

	dotProduct<<<blocks, threadsPerBlock>>>(A, B, C, N);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(err));
	}
}

int main() {
	int N = 1000;
	size_t bytes = N * sizeof(float);

	float *h_A = new float[N];
	float *h_B = new float[N];
	float h_C;

	for (int i = 0; i < N; i++) {
		h_A[i] = static_cast<float>(rand()) / RAND_MAX;
		h_B[i] = static_cast<float>(rand()) / RAND_MAX;
	}

	float *d_A, *d_B, *d_C;

	hipMalloc((void **)&d_A, bytes);
	hipMalloc((void **)&d_B, bytes);
	hipMalloc(&d_C, sizeof(float));

	hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
	
	solve(d_A, d_B, d_C, N);

	hipMemcpy(&h_C, d_C, sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);

	return 0;
}
