#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

extern "C" void solve(const float* A, const float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t bytes = N * sizeof(float);

    hipMalloc((void **)&d_A, bytes);
    hipMalloc((void **)&d_B, bytes);
    hipMalloc((void **)&d_C, bytes);

    hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
