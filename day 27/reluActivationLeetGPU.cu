#include <hip/hip_runtime.h>
#include <math.h>

__global__ void reluActivation(const float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}

extern "C" void solve(const float* input, float* output, int N) {
    float *d_input, *d_output;
    size_t bytes = N * sizeof(float);

    hipMalloc((void **)&d_input, bytes);
    hipMalloc((void **)&d_output, bytes);

    hipMemcpy(d_input, input, bytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    reluActivation<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, bytes, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
