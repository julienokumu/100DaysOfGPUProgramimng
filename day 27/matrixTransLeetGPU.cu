#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void matrixTranspose(const float* input, float* output, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int input_idx = row * cols + col;
        int output_idx = col * rows + row;

        output[output_idx] = input[input_idx];
    }
}

extern "C" void solve(const float* input, float* output, int rows, int cols) {
    float *d_input, *d_output;
    size_t bytesInput = rows * cols * sizeof(float);
    size_t bytesOutput = cols * rows * sizeof(float);

    hipMalloc((void **)&d_input, bytesInput);
    hipMalloc((void **)&d_output, bytesOutput);

    hipMemcpy(d_input, input, bytesInput, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE, (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrixTranspose<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, rows, cols);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, bytesOutput, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
