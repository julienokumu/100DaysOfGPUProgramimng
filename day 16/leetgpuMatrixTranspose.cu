#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void matrixTranspose(const float* input, float* output, int rows, int cols) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < rows && col < cols) {
		int input_idx = row * cols + col;
		int output_idx = col * rows + row;

		output[output_idx] = input[input_idx];
	}
}

extern "C" void solve(const float* input, float* output, int rows, int cols) {
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocksPerGrid((rows + BLOCK_SIZE - 1) / BLOCK_SIZE,
											(cols + BLOCK_SIZE - 1) / BLOCKS_SIZE);

	matrixTranspose<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(err));
	}
}

int main() {
	int rows = 2;
	int cols = 3;
	
	float h_input[] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0};
	float* h_output = (float*)malloc(cols * rows * sizeof(float));

	float *d_input, *d_output;
	hipMalloc((void **)&d_input, rows * cols * sizeof(float));
	hipMalloc((void **)&d_output, cols * rows * sizeof(float));

	hipMemcpy(d_input, h_input, rows * cols * sizeof(float), hipMemcpyHostToDevice);
	
	solve(d_input, d_output, rows, cols);

	hipMemcpy(h_output, d_output, cols * rows * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_input);
	hipFree(d_output);
	free(h_input);

	return 0;
}
