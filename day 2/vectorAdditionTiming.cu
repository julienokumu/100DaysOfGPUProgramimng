#include <stdio.h>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	for (; idx < size; idx += blockDim.x * gridDim.x) {
		C[idx] = A[idx] + B[idx];
	}
}

void vectorAddCPU(const float *A, const float *B, float *C, int size) {
	for (int i = 0; i < size; i++) {
		C[i] = A[i] + B[i];
	}
}

int main() {
	const int N = 1000000;
	size_t size = N * sizeof(float);

	float *h_A = (float *)malloc(size);
	float *h_B = (float *)malloc(size);
	float *h_C = (float *)malloc(size);
	float *h_C_cpu = (float *)malloc(size);

	for (int i = 0; i < N; i++) {
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);

	int threadsPerBlock = 256;
	int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
	vectorAdd<<<blocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(err));

		return 1;
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float gpu_time = 0;
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("GPU execution time: %.3f ms\n", gpu_time);

	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	auto cpu_start = std::chrono::high_resolution_clock::now();
	vectorAddCPU(h_A, h_B, h_C_cpu, N);
	auto cpu_end = std::chrono::high_resolution_clock::now();
	
	float cpu_time = std::chrono::duration<float, std::milli>(cpu_end - cpu_start).count();
	printf("CPU execution time: %.3f ms\n", cpu_time);

	bool correct = true;
	for (int i = 0; i < N; i++) {
		if (fabs(h_C[i] - h_C_cpu[i]) > 1e-5) {
			printf("verification failed at index %d: GPU = %.2f, CPU = %.2f\n", i, h_C[i], h_C_cpu[i]);

			correct = false;
			break;
		}
	}

	if (correct) {
		printf("vector addition completed successfully\n");
		printf("speedup: %.2fX\n", cpu_time / gpu_time);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_cpu);

	return 0;
}
