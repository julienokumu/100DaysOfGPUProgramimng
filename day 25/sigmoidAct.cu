#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void sigmoidAct(const float* x, float* y, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N) {
		y[idx] = 1.0f / (1.0f + expf(-x[idx]));
	}
}

void solve(const float* x, float* y, int N) {
	const int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	sigmoidAct<<<blocksPerGrid, threadsPerBlock>>>(x, y, N);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: ", hipGetErrorString(err));
	}
}

int main() {
	int N = 1 << 20;
	size_t bytes = N * sizeof(float);

	float* h_x = new float[N];
	float* h_y = new float[N];

	for (int i = 0; i < N; i++) {
		h_x[i] = -0.5f + 10.0f * static_cast<float>(rand()) / RAND_MAX;
	}

	float *d_x, *d_y;
	hipMalloc((void **)&d_x, bytes);
	hipMalloc((void **)&d_y, bytes);

	hipMemcpy(d_x, h_x, bytes, hipMemcpyHostToDevice);

	solve(d_x, d_y, N);

	hipMemcpy(h_y, d_y, bytes, hipMemcpyDeviceToHost);

	printf("First 5 sigmoid outputs: \n");
	for (int i = 0; i < 5 && i < N; i++) {
		printf("y[%d] = %f (x[%d] = %f)\n", i, h_y[i], i, h_x[i]);
	}

	hipFree(d_x);
	hipFree(d_y);
	free(h_x);
	free(h_y);

	return 0;
}

