#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixAdd(const float *A, const float *B, float *C, int cols, int rows) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int stride_x = blockDim.x * gridDim.x;
	int stride_y = blockDim.y * gridDim.y;

	for (; row < rows; row += stride_y) {
		for (int j = col; j < cols; j += stride_x) {
			int idx =  row * cols + j;
			C[idx] = A[idx] + B[idx];
		}
	}
}

int main() {
	const int ROWS = 100;
	const int COLS = 100;
	size_t size = ROWS * COLS * sizeof(float);
	const dim3 blockSizes[] = {dim3(16, 16), dim3(32, 32)};
	const int numTests = 2;

	float *h_A = (float *)malloc(size);
	float *h_B = (float *)malloc(size);
	float *h_C = (float *)malloc(size);

	for (int i = 0; i < ROWS * COLS; i++) {
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	for (int t = 0; t < numTests; t++) {
		dim3 blockSize = blockSizes[t];
		dim3 gridSize((COLS + blockSize.x - 1) / blockSize.x,
									(ROWS + blockSize.y - 1) / blockSize.y);
		printf("\nTesting block size: %dx%d (grid: %d%d)\n", blockSize.x, blockSize.y, gridSize.x, gridSize.y);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);

		matrixAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, ROWS, COLS);

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			printf("CUDA error: %s\n", hipGetErrorString(err));

			return 1;
		}

		hipEventRecord(stop);
		hipEventSynchronize(stop);

		float gpu_time = 0;
		hipEventElapsedTime(&gpu_time, start, stop);
		printf("GPU execution time: %.3f ms\n", gpu_time);

		hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

		printf("first 5 results (row 0):\n");
		for (int j = 0; j < 5; j++) {
			int idx = 0 * COLS + j;
			printf("C[0][%d] = %.2f (A[0][%d] = %.2f + B[0][%d] = %.2f)\n", j, h_C[idx], j, h_A[idx], j, h_C[idx]);
		}

		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
