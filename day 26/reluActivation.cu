#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void reluActivation(const float* x, float* y, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N) {
		y[idx] = fmaxf(0.0f, x[idx]);
	}
}

void solutions(const float* x, float* y, int N) {
	const int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	reluActivation<<<blocksPerGrid, threadsPerBlock>>>(x, y, N);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: ", hipGetErrorString(err));
	}
}

int main() {
	int N = 1 << 20;
	size_t bytes = N * sizeof(float);

	float* h_x = new float[N];
	float* h_y = new float[N];

	for (int i = 0; i < N; i++) {
		h_x[i] = -5.0f + 10.0f * static_cast<float>(rand()) / RAND_MAX;
	}

	float *d_x, *d_y;
	hipMalloc((void **)&d_x, bytes);
	hipMalloc((void **)&d_y, bytes);

	hipMemcpy(d_x, h_x, bytes, hipMemcpyHostToDevice);

	solution(d_x, d_y, N);

	hipMemcpy(h_y, d_y, bytes, hipMemcpyDeviceToHost);

	printf("First 5 ReLU outputs: \n");
	for (int i = 0; i < 5 && i < N; i++) {
		printf("y[%d] = %f (x[%d] = %f)\n", i, h_y[i], i, h_x[i]);
	}

	hipFree(d_x);
	hipFree(d_y);
	free(h_x);
	free(h_y);

	return 0;
}
