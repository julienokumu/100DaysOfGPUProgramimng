#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void L2NormKernel(const float* input, float* output, float* globalSum, int N) {
	__shared__ float sharedMem[256];
	
	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	float acc = 0.0f;
	if (idx < N) {
		acc = input[idx] * input[idx];
	}
	sharedMem[tid] = acc;
	__syncthreads();

	for (int stride = blockDim.x / 2; stride > 0; stride >>=1) {
		if (tid < stride) {
			sharedMem[tid] += sharedMem[tid + stride];
		}
		__syncthreads();
	}

	if (tid == 0) {
		atomicAdd(globalSum, sharedMem[0]);
	}
	__syncthreads();

	volatile float *norm = globalSum;
	float L2Norm = sqrtf(*norm);

	if (idx < N && L2Norm > 0.0f) {
			output[idx] = input[idx] / L2Norm;
	}
}

	void solve(const float* input, float* output, int N) {
		const int threadsPerBlock = 256;
		int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	
		float* d_globalSum;
		hipMemset(d_globalSum, 0, sizeof(float));
		hipMalloc((void **)&d_globalSum, sizeof(float));

		L2NormKernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, d_globalSum, N);
		hipDeviceSynchronize();

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			printf("CUDA error: ", hipGetErrorString(err));
		}

		hipFree(d_globalSum);
	}

	int main() {
		int N = 1 << 20;
		size_t bytes = N * sizeof(float);

		float* h_input = new float[N];
		float* h_output = new float[N];

		for (int i = 0; i < N; i++) {
			h_input[i] = static_cast<float>(rand()) / RAND_MAX;
		}

		float *d_input, *d_output;
		hipMalloc((void **)&d_input, bytes);
		hipMalloc((void **)&d_output, bytes);

		hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

		solve(d_input, d_output, N);

		hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost);

		printf("First 5 normalized values:\n");
		for (int i = 0; i < 5 && i < N; i++) {
			printf("output[%d] = %f (input[%d] = %f)\n", i, h_output[i], i, h_input[i]);
		}

		hipFree(d_input);
		hipFree(d_output);
		free(h_input);
		free(h_output);

		return 0;
	}
 
