#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void GEMM(const float* A, const float* B, float* C, int m, int n, int k) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < m && col < n) {
		float sum = 0.0f;

		for (int l = 0; l < k; l++) {
			sum += A[row * k + l] * B[l * n + col];
		}

		C[row * n + col] = sum;
	}
}

void solve(const float* A, const float* B, float* C, int m, int n, int k) {
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);

	GEMM<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, m, n, k);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(err));
	}
}

int main() {
	int m = 256;
	int n = 128;
	int k = 512;

	float* h_A = new float[m * k];
	float* h_B = new float[k * n];
	float* h_C = new float[m * n];

	for (int i = 0; i < m; i++) {
		for (int j = 0; j < k; j++) {
			h_A[i * k + j] = static_cast<float>(i + j + 1);
		}
	}

	for (int i = 0; i < k; i++) {
		for (int j = 0; j < n; j++) {
			h_B[i * n + j] = static_cast<float>(1.0 / (i + j + 1));
		}
	}

	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, m * k * sizeof(float));
	hipMalloc((void **)&d_B, k * n * sizeof(float));
	hipMalloc((void **)&d_C, m * n * sizeof(float));

	hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice);

	solve(d_A, d_B, d_C, m, n, k);

	hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

	printf("Sample results: \n");
	for (int i = 0; i < 5 && i < m * n; i++) {
		int row = i / n;
		int col = i % n;
		printf("C[d%][%d] = %f\n", row, col, h_C[i]);
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}

