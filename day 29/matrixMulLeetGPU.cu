#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void matrixMul(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < K) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * K + col];
        }
        C[row * K + col] = sum;
    }
}

extern "C" void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((K + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
