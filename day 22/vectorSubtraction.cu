#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorSub(const float* A, const float* B, float* C, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N) {
		C[idx] = A[idx] - B[idx];
	}
}

void solve(const float* A, const float* B, float* C, int N) {
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	vectorSub<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: ", hipGetErrorString(err));
	}
}

int main() {
	int N = 1 << 20;
	size_t bytes = N * sizeof(float);

	float* h_A = new float[N];
	float* h_B = new float[N];
	float* h_C = new float[N];

	for (int i = 0; i < N; i++) {
		h_A[i] = static_cast<float>(i + 1);
		h_B[i] = static_cast<float>(i * 0.5f);
	}

	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, bytes);
	hipMalloc((void **)&d_B, bytes);
	hipMalloc((void **)&d_C, bytes);

	hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

	solve(d_A, d_B, d_C, N);

	hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

	printf("Sample results (first 5 elements):\n");
  for (int i = 0; i < 5 && i < N; i++) {
      printf("c[%d] = %f (a[%d] = %f, b[%d] = %f)\n", i, h_C[i], i, h_A[i], i, h_B[i]);
  }

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	delete[] h_A;
	delete[] h_B;
	delete[] h_C;

	return 0;
}
