#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void arrayMulOptimized(const float *A, const float *B, float *C, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N) {
		C[idx] = A[idx] * B[idx];
	}
}

int main() {
	const int N = 10000;
	size_t bytes = N * sizeof(float);

	const int blockSizes[] = {64, 128, 256};
	const int numTests = 3;

	float *h_A = (float *)malloc(bytes);
	float *h_B = (float *)malloc(bytes);
	float *h_C = (float *)malloc(bytes);

	for (int i = 0; i < N; i++) {
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, bytes);
	hipMalloc((void **)&d_B, bytes);
	hipMalloc((void **)&d_C, bytes);

	hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

	for (int t = 0; t < numTests; t++) {
		int threadsPerBlock = blockSizes[t];
		int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

		printf("\nTesting block size: %d (blocks: %d)\n", threadsPerBlock, blocks);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		
		hipEventRecord(start);

		arrayMulOptimized<<<blocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

		hipEventRecord(stop);
		hipEventSynchronize(stop);

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			printf("CUDA error: %s\n", hipGetErrorString(err));
			
			return 1;
		}

		float gpu_time = 0;
		hipEventElapsedTime(&gpu_time, start, stop);
		printf("GPU execution time: %.3f ms\n", gpu_time);

		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
