#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>

__global__ void maxReduction(const float* input, float* output, int N) {
	extern __shared__ float sharedMem[];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;

	sharedMem[tid] = (idx < N) ? input[idx] : -FLT_MAX;
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>=1) {
		if (tid < s) {
			sharedMem[tid] = fmaxf(sharedMem[tid], sharedMem[tid + s]);
		}
		__syncthreads();
	}

	if (tid == 0) {
		*output = sharedMem[0];
	}
}

int main() {
	const int N = 1024;
	size_t bytes = N * sizeof(float);
	srand(42);

	float* h_input = (float *)malloc(bytes);
	for (int i = 0; i < N; i++) {
		h_input[i] = (rand() / (float)RAND_MAX) * 100.0f;
	}

	float* d_input, *d_output;
	hipMalloc((void **)&d_input, bytes);
	hipMalloc((void **)*&d_output, bytes);

	hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

	float h_output = -FLT_MAX;
	hipMemcpy(d_output, &h_output, bytes, hipMemcpyHostToDevice);

	int threadsPerBlock = 512;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	size_t sharedMemSize = threadsPerBlock * sizeof(float);

	maxReduction<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_input, d_output, N);
	hipDeviceSynchronize();

	hipMemcpy(&h_output, d_output, bytes, hipMemcpyDeviceToHost);

	printf("Max value: %f\n", h_output);

	hipFree(d_input);
	hipFree(d_output);
	free(h_input);

	return 0;
}
