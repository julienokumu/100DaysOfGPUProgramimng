#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dotProduct(const float* A, const float* B, float* result, int N) {
	extern __shared__ float sharedMem[];

	unsigned int tid = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	float sum = 0.0f;

	while (idx < N) {
		sum += A[idx] * B[idx];
		idx += blockDim.x * gridDim.x;
	}

	sharedMem[tid] = sum;
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s >>=1) {
		if (tid < s) {
			sharedMem[tid] += sharedMem[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) {
		atomicAdd(result, sharedMem[0]);
	}
}

extern "C" void solve(const float* A, const float* B, float* result, int N) {
	hipMemset(result, 0, sizeof(float));

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	size_t sharedMemSize = threadsPerBlock * sizeof(float);

	dotProduct<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(A, B, result, N);
	hipDeviceSynchronize();
}

int main() {
	int N = 4;
	size_t = bytes = N * sizeof(float);

	float h_A[] = {1.0, 2.0, 3.0, 4.0};
	float h_B[] = {5.0, 6.0, 7.0, 8.0};
	float h_result = 0.0f;

	float *d_A, *d_B, *d_result;
	hipMalloc((void **)&d_A, bytes);
	hipMalloc((void **)&d_B, bytes);
	hipMalloc((void **)&d_C, sizeof(float));

	hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

	solve(d_A, d_B, d_result, N);

	hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

	printf("Dot Product: %.1f\n", h_result);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_result);
	free(h_A);
	free(h_B);

	return 0;
}
